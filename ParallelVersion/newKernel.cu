#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "md5.cu"
/*
 *   Maximum number of threads per multiprocessor:  2048
 *   Maximum number of threads per block:           1024
 *   Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
 *   Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
 *
 */
#define MAX_GRID_X 2147483647
#define MAX_GRID_Y 65535
#define MAX_GRID_z 65535

#define MAX_BLOCK_X 1024
#define MAX_BLOCK_Y 1024
#define MAX_BLOCK_Z 64

__global__ void crack(uint wordLength, uint beginningOffset, long long batchSize, unsigned char *out, unsigned char *charMap, uint charSetLength, uint v1, uint v2, uint v3, uint v4){
    long long permutationNo = gridDim.x * blockIdx.y + blockIdx.x;

    extern __shared__ unsigned char thisWord[];
    if(permutationNo > batchSize)
        return;

    //permutationNo += beginningOffset;

    //int thisValue = permutationNo % (charSetLength * (threadIdx.x + 1) + 1);
    //thisWord[threadIdx.x] = charMap[thisValue];
    //uint c1,c2,c3,c4;
    //md5_vfy(thisWord, wordLength, &c1, &c2, &c3, &c4);

    //if(c1 == v1 && c2 == v2 && c3 == v3 && c4 == v4 ){
    //    out[threadIdx.x] = thisWord[threadIdx.x];
   // }
}

void usage(char* programName);

int main(int argc, char** argv){
    hipDeviceReset();
    // Device
    unsigned char *d_charMap, *d_out;

    // Host
    unsigned char *h_charMap, *h_out;
    uint h_wordLength, h_batchSize, h_charSetLength, v1, v2, v3, v4;
    int inputWordLength;
    int charMapLength;


    // Configuration variables
    dim3 gridDim;
    dim3 blockDim;


    if(argc < 2)
        usage(argv[0]);

    unsigned char* inputWord = (unsigned char*) calloc(strlen(argv[1]) + 1, sizeof(unsigned char));
    strcpy((char*)inputWord, argv[1]);
    inputWordLength = strlen((const char*)inputWord);


    // Generate hash
    md5_vfy(inputWord, inputWordLength, &v1, &v2, &v3, &v4);

    // Allocate cpu memory
    char* staticCharSet = (char*)"abcdefghijklmnopqrstuvwxyz";
    charMapLength = strlen((char*)staticCharSet);
    h_charMap = (unsigned char*) calloc(charMapLength,   sizeof(unsigned char));
    h_out     = (unsigned char*) calloc(inputWordLength, sizeof(unsigned char));
    strcpy((char*)h_charMap, (const char*) staticCharSet);


    // Allocate and initialize Gpu memory
    hipMalloc((void **) &d_charMap, sizeof(unsigned char) * charMapLength);
    hipMalloc((void **) &d_out,     sizeof(unsigned char) * inputWordLength);
    hipMemset (d_charMap,0,sizeof(unsigned char) * charMapLength);
    hipMemset (d_out,0,sizeof(unsigned char) * inputWordLength);
    hipMemcpy(d_charMap, h_charMap, charMapLength * sizeof(unsigned char), hipMemcpyHostToDevice);


    // Calculate the number of possible permutations
    int digitNo = 1;
    long long noPermutations = charMapLength;
    for(; digitNo < inputWordLength; ++digitNo){
        noPermutations *= charMapLength;
    }

//    blockDim.x = testWordLength;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x  = min(noPermutations, MAX_GRID_Y) ;//(int) noPermutations;//ceil(MAX_GRID_X / testWordLength);
    gridDim.y  = ceil(noPermutations / gridDim.x);
    gridDim.z  = 1;

    printf("Input Word: %s\nInput Word Length: %d\nCharacter Set:\"%s\"\nPossible Permutations: %d\n", inputWord, inputWordLength, h_charMap, noPermutations);
    int testWordLength = 1;
    for(; testWordLength <= inputWordLength; ++testWordLength){
        blockDim.x = testWordLength;
        crack <<< gridDim, blockDim, testWordLength >>> (testWordLength, 0, noPermutations, d_out, d_charMap, charMapLength, v1, v2, v3, v4);
        hipMemcpy(h_out, d_out, testWordLength, hipMemcpyDeviceToHost);
        if(h_out[0] != '\0'){
            printf("Found match: %s\n", h_out);
            break;
        }

    }
    if(h_out[0] == '\0')
        printf("No match was found :(\n");


    hipFree(d_charMap);
    hipFree(d_out);
}


void usage(char* programName){
    fprintf(stderr, "usage: %s testWord\n", programName);
    exit(1);
}